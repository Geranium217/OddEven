#include <hip/hip_runtime.h> #include <stdio.h> #include <stdlib.h>

#define BLOCK_SIZE 256

global void oddEvenMergeSort(int* arr, int n) { int idx = blockIdx.x * blockDim.x + threadIdx.x; if (idx < n) { if (idx % 2 == 0) { // odd index, compare with next element if (arr[idx] > arr[idx + 1]) { int temp = arr[idx]; arr[idx] = arr[idx + 1]; arr[idx + 1] = temp; } } else { // even index, compare with previous element if (idx > 0 && arr[idx] < arr[idx - 1]) { int temp = arr[idx]; arr[idx] = arr[idx - 1]; arr[idx - 1] = temp; } } } }

int main() { int n; printf("Enter the size of the array: "); scanf("%d", &n);

int* arr = (int*)malloc(n * sizeof(int));
printf("Enter the values to be sorted (separated by spaces): ");
for (int i = 0; i < n; i++) {
    scanf("%d", &arr[i]);
}

// copy array to device
int* d_arr;
hipMalloc((void**)&d_arr, n * sizeof(int));
hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

// launch kernel
int blockSize = BLOCK_SIZE;
int numBlocks = (n + blockSize - 1) / blockSize;
oddEvenMergeSort<<<numBlocks, blockSize>>>(d_arr, n);

// copy result back to host
int* h_arr = (int*)malloc(n * sizeof(int));
hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

// print sorted array
printf("Sorted array: ");
for (int i = 0; i < n; i++) {
    printf("%d ", h_arr[i]);
}
printf("\n");

free(arr);
free(h_arr);
hipFree(d_arr);

return 0;

}
